#include "hip/hip_runtime.h"
/*
 * yolo_layer.cu
 *
 * This code was originally written by wang-xinyu under MIT license.
 * I took it from:
 *
 *     https://github.com/wang-xinyu/tensorrtx/tree/master/yolov4
 *
 * and made necessary modifications.
 *
 * - JK Jung
 */

#include "yolo_layer.h"

using namespace Yolo;

namespace
{
// Write values into buffer
template <typename T>
void write(char*& buffer, const T& val)
{
    *reinterpret_cast<T*>(buffer) = val;
    buffer += sizeof(T);
}

// Read values from buffer
template <typename T>
void read(const char*& buffer, T& val)
{
    val = *reinterpret_cast<const T*>(buffer);
    buffer += sizeof(T);
}
} // namespace

namespace nvinfer1
{
    YoloLayerPlugin::YoloLayerPlugin(int yolo_width, int yolo_height, int num_anchors, float* anchors, int num_classes, int input_width, int input_height, float scale_x_y, int new_coords)
    {
        mYoloWidth   = yolo_width;
        mYoloHeight  = yolo_height;
        mNumAnchors  = num_anchors;
        memcpy(mAnchorsHost, anchors, num_anchors * 2 * sizeof(float));
        mNumClasses  = num_classes;
        mInputWidth  = input_width;
        mInputHeight = input_height;
        mScaleXY     = scale_x_y;
        mNewCoords   = new_coords;

        CHECK(hipMalloc(&mAnchors, MAX_ANCHORS * 2 * sizeof(float)));
        CHECK(hipMemcpy(mAnchors, mAnchorsHost, mNumAnchors * 2 * sizeof(float), hipMemcpyHostToDevice));
    }

    YoloLayerPlugin::YoloLayerPlugin(const void* data, size_t length)
    {
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mThreadCount);
        read(d, mYoloWidth);
        read(d, mYoloHeight);
        read(d, mNumAnchors);
        memcpy(mAnchorsHost, d, MAX_ANCHORS * 2 * sizeof(float));
        d += MAX_ANCHORS * 2 * sizeof(float);
        read(d, mNumClasses);
        read(d, mInputWidth);
        read(d, mInputHeight);
        read(d, mScaleXY);
        read(d, mNewCoords);

        CHECK(hipMalloc(&mAnchors, MAX_ANCHORS * 2 * sizeof(float)));
        CHECK(hipMemcpy(mAnchors, mAnchorsHost, mNumAnchors * 2 * sizeof(float), hipMemcpyHostToDevice));

        assert(d == a + length);
    }

    void YoloLayerPlugin::serialize(void* buffer) const
    {
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mThreadCount);
        write(d, mYoloWidth);
        write(d, mYoloHeight);
        write(d, mNumAnchors);
        memcpy(d, mAnchorsHost, MAX_ANCHORS * 2 * sizeof(float));
        d += MAX_ANCHORS * 2 * sizeof(float);
        write(d, mNumClasses);
        write(d, mInputWidth);
        write(d, mInputHeight);
        write(d, mScaleXY);
        write(d, mNewCoords);

        assert(d == a + getSerializationSize());
    }

    size_t YoloLayerPlugin::getSerializationSize() const
    {
        return sizeof(mThreadCount) + \
               sizeof(mYoloWidth) + sizeof(mYoloHeight) + \
               sizeof(mNumAnchors) + MAX_ANCHORS * 2 * sizeof(float) + \
               sizeof(mNumClasses) + \
               sizeof(mInputWidth) + sizeof(mInputHeight) + \
               sizeof(mScaleXY) + sizeof(mNewCoords);
    }

    int YoloLayerPlugin::initialize()
    {
        return 0;
    }

    void YoloLayerPlugin::terminate()
    {
        CHECK(hipFree(mAnchors));
    }

    Dims YoloLayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
        assert(index == 0);
        assert(nbInputDims == 1);
        assert(inputs[0].d[0] == (mNumClasses + 5) * mNumAnchors);
        assert(inputs[0].d[1] == mYoloHeight);
        assert(inputs[0].d[2] == mYoloWidth);
        // output detection results to the channel dimension
        int totalsize = mYoloWidth * mYoloHeight * mNumAnchors * sizeof(Detection) / sizeof(float);
        return Dims3(totalsize, 1, 1);
    }

    void YoloLayerPlugin::setPluginNamespace(const char* pluginNamespace)
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* YoloLayerPlugin::getPluginNamespace() const
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType YoloLayerPlugin::getOutputDataType(int index, const DataType* inputTypes, int nbInputs) const
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool YoloLayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool YoloLayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const
    {
        return false;
    }

    void YoloLayerPlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput)
    {
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void YoloLayerPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)
    {
    }

    // Detach the plugin object from its execution context.
    void YoloLayerPlugin::detachFromContext()
    {
    }

    const char* YoloLayerPlugin::getPluginType() const
    {
        return "YoloLayer_TRT";
    }

    const char* YoloLayerPlugin::getPluginVersion() const
    {
        return "1";
    }

    void YoloLayerPlugin::destroy()
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* YoloLayerPlugin::clone() const
    {
        YoloLayerPlugin *p = new YoloLayerPlugin(mYoloWidth, mYoloHeight, mNumAnchors, (float*) mAnchorsHost, mNumClasses, mInputWidth, mInputHeight, mScaleXY, mNewCoords);
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    inline __device__ float sigmoidGPU(float x) { return 1.0f / (1.0f + __expf(-x)); }

    inline __device__ float scale_sigmoidGPU(float x, float s)
    {
        return s * sigmoidGPU(x) - (s - 1.0f) * 0.5f;
    }

    // CalDetection(): This kernel processes 1 yolo layer calculation.  It
    // distributes calculations so that 1 GPU thread would be responsible
    // for each grid/anchor combination.
    // NOTE: The output (x, y, w, h) are between 0.0 and 1.0
    //       (relative to orginal image width and height).
    __global__ void CalDetection(const float *input, float *output,
                                 int batch_size,
                                 int yolo_width, int yolo_height,
                                 int num_anchors, const float *anchors,
                                 int num_classes, int input_w, int input_h,
                                 float scale_x_y)
    {
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        Detection* det = ((Detection*) output) + idx;
        int total_grids = yolo_width * yolo_height;
        if (idx >= batch_size * total_grids * num_anchors) return;

        int info_len = 5 + num_classes;
        //int batch_idx = idx / (total_grids * num_anchors);
        int group_idx = idx / total_grids;
        int anchor_idx = group_idx % num_anchors;
        const float* cur_input = input + group_idx * (info_len * total_grids) + (idx % total_grids);

        int class_id;
        float max_cls_logit = -HIP_INF_F;  // minus infinity
        for (int i = 5; i < info_len; ++i) {
            float l = *(cur_input + i * total_grids);
            if (l > max_cls_logit) {
                max_cls_logit = l;
                class_id = i - 5;
            }
        }
        float max_cls_prob = sigmoidGPU(max_cls_logit);
        float box_prob = sigmoidGPU(*(cur_input + 4 * total_grids));
        //if (max_cls_prob < IGNORE_THRESH || box_prob < IGNORE_THRESH)
        //    return;

        int row = (idx % total_grids) / yolo_width;
        int col = (idx % total_grids) % yolo_width;

        det->bbox[0] = (col + scale_sigmoidGPU(*(cur_input + 0 * total_grids), scale_x_y)) / yolo_width;    // [0, 1]
        det->bbox[1] = (row + scale_sigmoidGPU(*(cur_input + 1 * total_grids), scale_x_y)) / yolo_height;   // [0, 1]
        det->bbox[2] = __expf(*(cur_input + 2 * total_grids)) * *(anchors + 2 * anchor_idx + 0) / input_w;  // [0, 1]
        det->bbox[3] = __expf(*(cur_input + 3 * total_grids)) * *(anchors + 2 * anchor_idx + 1) / input_h;  // [0, 1]

        det->bbox[0] -= det->bbox[2] / 2;  // shift from center to top-left
        det->bbox[1] -= det->bbox[3] / 2;

        det->det_confidence = box_prob;
        det->class_id = class_id;
        det->class_confidence = max_cls_prob;
    }

    inline __device__ float scale(float x, float s)
    {
        return s * x - (s - 1.0f) * 0.5f;
    }

    inline __device__ float square(float x)
    {
        return x * x;
    }

    __global__ void CalDetection_NewCoords(const float *input, float *output,
                                           int batch_size,
                                           int yolo_width, int yolo_height,
                                           int num_anchors, const float *anchors,
                                           int num_classes, int input_w, int input_h,
                                           float scale_x_y)
    {
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        Detection* det = ((Detection*) output) + idx;
        int total_grids = yolo_width * yolo_height;
        if (idx >= batch_size * total_grids * num_anchors) return;

        int info_len = 5 + num_classes;
        //int batch_idx = idx / (total_grids * num_anchors);
        int group_idx = idx / total_grids;
        int anchor_idx = group_idx % num_anchors;
        const float* cur_input = input + group_idx * (info_len * total_grids) + (idx % total_grids);

        int class_id;
        float max_cls_prob = -HIP_INF_F;  // minus infinity
        for (int i = 5; i < info_len; ++i) {
            float l = *(cur_input + i * total_grids);
            if (l > max_cls_prob) {
                max_cls_prob = l;
                class_id = i - 5;
            }
        }
        float box_prob = *(cur_input + 4 * total_grids);
        //if (max_cls_prob < IGNORE_THRESH || box_prob < IGNORE_THRESH)
        //    return;

        int row = (idx % total_grids) / yolo_width;
        int col = (idx % total_grids) % yolo_width;

        det->bbox[0] = (col + scale(*(cur_input + 0 * total_grids), scale_x_y)) / yolo_width;                   // [0, 1]
        det->bbox[1] = (row + scale(*(cur_input + 1 * total_grids), scale_x_y)) / yolo_height;                  // [0, 1]
        det->bbox[2] = square(*(cur_input + 2 * total_grids)) * 4 * *(anchors + 2 * anchor_idx + 0) / input_w;  // [0, 1]
        det->bbox[3] = square(*(cur_input + 3 * total_grids)) * 4 * *(anchors + 2 * anchor_idx + 1) / input_h;  // [0, 1]

        det->bbox[0] -= det->bbox[2] / 2;  // shift from center to top-left
        det->bbox[1] -= det->bbox[3] / 2;

        det->det_confidence = box_prob;
        det->class_id = class_id;
        det->class_confidence = max_cls_prob;
    }

    void YoloLayerPlugin::forwardGpu(const float* const* inputs, float* output, hipStream_t stream, int batchSize)
    {
        int num_elements = batchSize * mNumAnchors * mYoloWidth * mYoloHeight;

        //CHECK(hipMemset(output, 0, num_elements * sizeof(Detection)));

        if (mNewCoords) {
            CalDetection_NewCoords<<<(num_elements + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream>>>
                (inputs[0], output, batchSize, mYoloWidth, mYoloHeight, mNumAnchors, (const float*) mAnchors, mNumClasses, mInputWidth, mInputHeight, mScaleXY);
        } else {
            CalDetection<<<(num_elements + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream>>>
                (inputs[0], output, batchSize, mYoloWidth, mYoloHeight, mNumAnchors, (const float*) mAnchors, mNumClasses, mInputWidth, mInputHeight, mScaleXY);
        }
    }

    int YoloLayerPlugin::enqueue(int batchSize, const void* const* inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        forwardGpu((const float* const*)inputs, (float*)outputs[0], stream, batchSize);
        return 0;
    }

    YoloPluginCreator::YoloPluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* YoloPluginCreator::getPluginName() const
    {
        return "YoloLayer_TRT";
    }

    const char* YoloPluginCreator::getPluginVersion() const
    {
        return "1";
    }

    const PluginFieldCollection* YoloPluginCreator::getFieldNames()
    {
        return &mFC;
    }

    IPluginV2IOExt* YoloPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
    {
        assert(!strcmp(name, getPluginName()));
        const PluginField* fields = fc->fields;
        int yolo_width, yolo_height, num_anchors = 0;
        float anchors[MAX_ANCHORS * 2];
        int num_classes, input_multiplier, new_coords = 0;
        float scale_x_y = 1.0;

        for (int i = 0; i < fc->nbFields; ++i)
        {
            const char* attrName = fields[i].name;
            if (!strcmp(attrName, "yoloWidth"))
            {
                assert(fields[i].type == PluginFieldType::kINT32);
                yolo_width = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "yoloHeight"))
            {
                assert(fields[i].type == PluginFieldType::kINT32);
                yolo_height = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "numAnchors"))
            {
                assert(fields[i].type == PluginFieldType::kINT32);
                num_anchors = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "numClasses"))
            {
                assert(fields[i].type == PluginFieldType::kINT32);
                num_classes = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "inputMultiplier"))
            {
                assert(fields[i].type == PluginFieldType::kINT32);
                input_multiplier = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "anchors")){
                assert(num_anchors > 0 && num_anchors <= MAX_ANCHORS);
                assert(fields[i].type == PluginFieldType::kFLOAT32);
                memcpy(anchors, static_cast<const float*>(fields[i].data), num_anchors * 2 * sizeof(float));
            }
            else if (!strcmp(attrName, "scaleXY"))
            {
                assert(fields[i].type == PluginFieldType::kFLOAT32);
                scale_x_y = *(static_cast<const float*>(fields[i].data));
            }
            else if (!strcmp(attrName, "newCoords"))
            {
                assert(fields[i].type == PluginFieldType::kINT32);
                new_coords = *(static_cast<const int*>(fields[i].data));
            }
            else
            {
                std::cerr <<  "Unknown attribute: " << attrName << std::endl;
                assert(0);
            }
        }
        assert(yolo_width > 0 && yolo_height > 0);
        assert(anchors[0] > 0.0f && anchors[1] > 0.0f);
        assert(num_classes > 0);
        assert(input_multiplier == 8 || input_multiplier == 16 || input_multiplier == 32);
        assert(scale_x_y >= 1.0);

        YoloLayerPlugin* obj = new YoloLayerPlugin(yolo_width, yolo_height, num_anchors, anchors, num_classes, yolo_width * input_multiplier, yolo_height * input_multiplier, scale_x_y, new_coords);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* YoloPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
    {
        YoloLayerPlugin* obj = new YoloLayerPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    PluginFieldCollection YoloPluginCreator::mFC{};
    std::vector<PluginField> YoloPluginCreator::mPluginAttributes;
} // namespace nvinfer1
